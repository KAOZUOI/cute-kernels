#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>

int main(int argc, char** argv) {
    hipblasHandle_t handle;
    hipblasStatus_t status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("cuBLAS init failed\n");
        return -1;
    }

    hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);

    const int M = 4096;
    const int N = 4096;
    const int K = 4096;

    size_t A_size = M * K;
    size_t B_size = K * N;
    size_t C_size = M * N;
    
    half *h_A = (half*)malloc(A_size * sizeof(half));
    half *h_B = (half*)malloc(B_size * sizeof(half));
    half *h_C = (half*)malloc(C_size * sizeof(half));
    
    if (!h_A || !h_B || !h_C) {
        printf("memory alloc failed\n");
        return -1;
    }
    
    // 随机初始化矩阵
    for (int i = 0; i < A_size; i++) {
        h_A[i] = __float2half(static_cast<float>(rand()) / RAND_MAX);
    }
    
    for (int i = 0; i < B_size; i++) {
        h_B[i] = __float2half(static_cast<float>(rand()) / RAND_MAX);
    }
    
    half *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, A_size * sizeof(half));
    hipMalloc((void**)&d_B, B_size * sizeof(half));
    hipMalloc((void**)&d_C, C_size * sizeof(half));

    hipMemcpy(d_A, h_A, A_size * sizeof(half), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, B_size * sizeof(half), hipMemcpyHostToDevice);
    
    const __half alpha = __float2half(1.0f);
    const __half beta = __float2half(0.0f);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipStream_t stream;
    hipStreamCreate(&stream);
    hipblasSetStream(handle, stream);
    
    for (int i = 0; i < 10; i++) {
        status = hipblasGemmEx(handle,
                              HIPBLAS_OP_T,
                              HIPBLAS_OP_N,
                              N, M, K,
                              &alpha,
                              d_B, HIP_R_16F, K,
                              d_A, HIP_R_16F, K,
                              &beta,
                              d_C, HIP_R_16F, M,
                              HIP_R_32F,
                              CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        
        if (status != HIPBLAS_STATUS_SUCCESS) {
            printf("cuBLAS HGEMM failed during warmup: %d\n", status);
            return -1;
        }
    }
    
    hipStreamSynchronize(stream);
    
    const int num_iters = 20;
    float total_ms = 0.0f;
    float min_ms = 1e10f;
    float max_ms = 0.0f;

    for (int iter = 0; iter < num_iters; iter++) {
        hipEventRecord(start, stream);
        
        status = hipblasGemmEx(handle,
                              HIPBLAS_OP_T,
                              HIPBLAS_OP_N,
                              N, M, K,
                              &alpha,
                              d_B, HIP_R_16F, K,
                              d_A, HIP_R_16F, K,
                              &beta,
                              d_C, HIP_R_16F, M,
                              HIP_R_32F,
                              CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        
        hipEventRecord(stop, stream);
        hipEventSynchronize(stop);
        
        if (status != HIPBLAS_STATUS_SUCCESS) {
            printf("cuBLAS HGEMM failed: %d\n", status);
            return -1;
        }
        
        float ms = 0.0f;
        hipEventElapsedTime(&ms, start, stop);
        
        total_ms += ms;
        min_ms = min(min_ms, ms);
        max_ms = max(max_ms, ms);
    }
    
    float avg_ms = total_ms / num_iters;
    double flops = 2.0 * M * N * K;
    double avg_gflops = (flops * 1.0e-9) / (avg_ms * 1.0e-3);
    double peak_gflops = (flops * 1.0e-9) / (min_ms * 1.0e-3);
    
    printf("BLAS_GEMM:     [%6.1f]GFlop/s  (%6.4f)ms\n", avg_gflops, avg_ms);

    hipMemcpy(h_C, d_C, C_size * sizeof(half), hipMemcpyDeviceToHost);
    
    hipStreamDestroy(stream);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipblasDestroy(handle);
    
    return 0;
}